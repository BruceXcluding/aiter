// SPDX-License-Identifier: MIT
// Copyright (c) 2024, Advanced Micro Devices, Inc. All rights reserved.
#include <torch/all.h>
#include <ATen/cuda/HIPContext.h>
#include "py_itfs_common.h"

#include "fused_moe.hpp"
#include "ck/ck.hpp"
#include "ck/tensor_operation/gpu/device/gemm_specialization.hpp"
#include "ck/tensor_operation/gpu/device/impl/device_gemm_multiple_d_xdl_cshuffle_v3_b_preshuffle.hpp"
#include "ck/tensor_operation/gpu/device/impl/device_gemm_multiple_d_xdl_cshuffle_v3.hpp"
#include "ck/tensor_operation/gpu/element/element_wise_operation.hpp"
#include "ck/tensor_operation/gpu/element/unary_element_wise_operation.hpp"

#include "ck/library/utility/device_memory.hpp"
#include "ck/library/utility/host_tensor.hpp"
#include "ck/library/utility/host_tensor_generator.hpp"
#include "ck/library/utility/literals.hpp"
#include "ck/library/reference_tensor_operation/cpu/reference_moe_gemm.hpp"
#include "ck/library/utility/check_err.hpp"

#include "ck/utility/blkgemmpipe_scheduler.hpp"

torch::Tensor ck_moe(torch::Tensor &hidden_states,          // [m, k], input token
                     torch::Tensor &w1,                     // [e, n, k]/[e, 2*n, k], pre-shuffle([e, nr, kr, w])
                     torch::Tensor &w2,                     // [e, n, k], pre-shuffle([e, nr, kr, w])
                     torch::Tensor &topk_weights,           // [tokens, topk]
                     torch::Tensor &topk_ids,               // [tokens, topk]
                     std::optional<torch::Tensor> w1_scale, // [e, 1, n], gate(up) scale
                     std::optional<torch::Tensor> w2_scale, // [e, 1, k], down scale
                     std::optional<torch::Tensor> a1_scale, // [m, 1], token scale
                     std::optional<torch::Tensor> a2_scale, // [e, 1, n], smooth-quant-scale for 2nd gemm input
                     std::optional<int> block_m = 32)
{
    auto device = hidden_states.device();
    int topk_ids_numel = topk_ids.numel();
    int experts = w1.size(0);
    int topk = topk_ids.size(1);
    int tokens = topk_ids.size(0);
    int hidden_size = w1.size(2);
    int shared_intermediate_size_0 = w1.size(1);
    int shared_intermediate_size = w2.size(-1);
    int block_size = block_m.value();

    int max_num_tokens_padded = topk_ids_numel + experts * block_size - topk;
    int max_num_m_blocks = (max_num_tokens_padded + block_size - 1) / block_size;

    auto sorted_ids = torch::empty({max_num_tokens_padded}, torch::TensorOptions().dtype(torch::kInt32).device(device));
    auto sorted_weights = torch::empty({max_num_tokens_padded}, torch::TensorOptions().dtype(torch::kFloat32).device(device));
    auto sorted_expert_ids = torch::empty({max_num_m_blocks}, torch::TensorOptions().dtype(torch::kInt32).device(device));
    auto num_tokens_post_pad = torch::empty({1}, torch::TensorOptions().dtype(torch::kInt32).device(device));
    auto out = torch::empty({tokens, hidden_size}, torch::TensorOptions().dtype(hidden_states.dtype()).device(device));

    auto prec_i = torchDTypeToStr(hidden_states.dtype());
    auto prec_w = torchDTypeToStr(w1.dtype());
    auto prec_o = torchDTypeToStr(out.dtype());
    auto prec_kw = torchDTypeToStr(topk_weights.dtype());

    int gate_only = 1;
    int activation = 0;
    int fused_quant = 0;
    if (shared_intermediate_size_0 == 2 * shared_intermediate_size)
    {
        gate_only = 0;
        activation = 1;
    }

    if (!w1_scale.has_value())
    {
        fused_quant = 0;
    }
    else if (a1_scale.has_value() && a2_scale.has_value())
    {
        fused_quant = 1;
    }
    else
    {
        fused_quant = 2;
    }

    int stride = hidden_size;
    std::string prec_st = !a1_scale ? "fp32" : torchDTypeToStr(a1_scale->dtype());
    std::string prec_sw = !w1_scale ? "fp32" : torchDTypeToStr(w1_scale->dtype());
    std::string prec_sq = !a2_scale ? "fp32" : torchDTypeToStr(a2_scale->dtype());

    fused_moe_traits traits{prec_i,
                            prec_w,
                            prec_o,
                            prec_st,
                            prec_sw,
                            prec_sq,
                            prec_kw,
                            block_size,
                            // activation, //need this when back to main branch
                            gate_only,
                            fused_quant};

    fused_moe_args args{hidden_states.data_ptr(),
                        a1_scale.has_value() ? a1_scale.value().data_ptr() : nullptr,
                        w1.data_ptr(),
                        w2.data_ptr(),
                        w1_scale.has_value() ? w1_scale.value().data_ptr() : nullptr,
                        w2_scale.has_value() ? w2_scale.value().data_ptr() : nullptr,
                        a2_scale.has_value() ? a2_scale.value().data_ptr() : nullptr,
                        out.data_ptr(),

                        topk_ids.data_ptr(),
                        topk_weights.data_ptr(),
                        sorted_ids.data_ptr(),
                        sorted_weights.data_ptr(),
                        sorted_expert_ids.data_ptr(),
                        num_tokens_post_pad.data_ptr(),

                        block_size,
                        hidden_size,
                        shared_intermediate_size,
                        tokens,
                        experts,
                        topk,
                        stride};

    const hipStream_t stream = at::cuda::getCurrentCUDAStream();

    fused_moe(traits, args, {stream});
    return out;
}

template <ck::index_t... Is>
using S = ck::Sequence<Is...>;
using F16 = ck::half_t;
using F32 = float;

using Row = ck::tensor_layout::gemm::RowMajor;
using Col = ck::tensor_layout::gemm::ColumnMajor;

using A0DataType = F16;
using B0DataType = F16;
using AccDataType = F32;
using CShuffleDataType = F32;
using D0DataType = F32;
using D1DataType = F32;
using DsDataType = ck::Tuple<D0DataType, D1DataType>;
using EDataType = F16;

using A0Layout = Row;
using B0Layout = Col;
using D0Layout = Row;
using D1Layout = Col;
using DsLayout = ck::Tuple<D0Layout, D1Layout>;
using ELayout = Row;
struct MultiplyMultiply
{
    template <typename E, typename C, typename D0, typename D1>
    __host__ __device__ constexpr void
    operator()(E &e, const C &c, const D0 &d0, const D1 &d1) const;

    template <>
    __host__ __device__ constexpr void operator()<F16, float, float, float>(F16 &e,
                                                                            const float &c,
                                                                            const float &d0,
                                                                            const float &d1) const
    {
        // const float x0_f = c * d0 * d1;
        const float x0_f = c;
        // printf("epi %f\n", c);
        e = ck::type_convert<F16>(x0_f);
    }

    // template <>
    // __host__ __device__ constexpr void operator()<BF16, float, float, float>(BF16& e,
    //                                                                          const float& c,
    //                                                                          const float& d0,
    //                                                                          const float& d1) const
    // {
    //     const float x0_f = c;
    //     // const float x0_f = c * d0 * d1;

    //     e = ck::type_convert<BF16>(x0_f);
    // }
};

void ck_moe_stage1(torch::Tensor &hidden_states,          // [m, k], input token
                   torch::Tensor &w1,                     // [e, n, k]/[e, 2*n, k], pre-shuffle([e, nr, kr, w])
                   torch::Tensor &w2,                     // [expert, dim, inter_dim], pre-shuffle([e, nr, kr, w])
                   torch::Tensor &sorted_token_ids,       // [max_num_tokens_padded]
                   torch::Tensor &sorted_expert_ids,      // [max_num_m_blocks]
                   torch::Tensor &out,                    // [max_num_tokens_padded, inter_dim]
                   std::optional<torch::Tensor> w1_scale, // [e, 1, n], gate(up) scale
                   std::optional<torch::Tensor> a1_scale  // [m, 1], token scale
)
{
    int tokens = hidden_states.size(0);
    int SORTED_SIZE = out.size(0);
    int N = w2.size(2);
    int K = w1.size(2);

    // ~~~~~~~~~~~~~~~~~~~~~~~~following start with ck things
    ck::index_t StrideA = K;
    ck::index_t StrideB = K;
    ck::index_t StrideD = 0;
    ck::index_t StrideE = N;
    ck::index_t KBatch = 1;
    using PassThrough = ck::tensor_operation::element_wise::PassThrough;

    using AElementOp = PassThrough;
    using BElementOp = PassThrough;
    using CDEElementOp = MultiplyMultiply;

    static constexpr auto GemmSpec = ck::tensor_operation::device::GemmSpecialization::Default;

    // using DeviceOpInstance = ck::tensor_operation::device::DeviceGemmMultiD_Xdl_CShuffle_V3
    using DeviceOpInstance = ck::tensor_operation::device::DeviceGemmMultiD_Xdl_CShuffle_V3_BPreshuffle
        // clang-format off
///######|  ALayout|  BLayout| DsLayout| ELayout|      AData|      BData|     DsData|     EData|     AccData|         CShuffle|           A|           B|          CDE|           GEMM| Block|  MPer|  NPer|  KPer| AK1| BK1| MPer| NPer| MXdl| NXdl|  ABlockTransfer| ABlockTransfer| ABlockTransfer| ABlockTransfer| ABlockTransfer| ABlockTransfer| ABlockLds|  BBlockTransfer| BBlockTransfer| BBlockTransfer| BlockTransfer| BBlockTransfer| BBlockTransfer| BBlockLds|    CShuffle|    CShuffle| CBlockTransferClusterLengths|  CBlockTransfer|
///######|         |         |         |        |       Type|       Type|       Type|      Type|        Type|         DataType| Elementwise| Elementwise|  Elementwise| Spacialization|  Size| Block| Block| Block|    |    |  XDL|  XDL|  Per|  Per|   ThreadCluster|  ThreadCluster| SrcAccessOrder|   SrcVectorDim|      SrcScalar|      DstScalar| AddExtraM|   ThreadCluster|  ThreadCluster| SrcAccessOrder|  SrcVectorDim|      SrcScalar|      DstScalar| AddExtraN| MXdlPerWave| NXdlPerWave|         _MBlock_MWaveMPerXdl| ScalarPerVector|
///######|         |         |         |        |           |           |           |          |            |                 |   Operation|   Operation|    Operation|               |      |      |      |      |    |    |     |     | Wave| Wave| Lengths_K0_M_K1|   ArrangeOrder|               |               |      PerVector|   PerVector_K1|          | Lengths_K0_N_K1|   ArrangeOrder|               |              |      PerVector|   PerVector_K1|          |  PerShuffle|  PerShuffle|         _NBlock_NWaveNPerXdl|   _NWaveNPerXdl|
///######|         |         |         |        |           |           |           |          |            |                 |            |            |             |               |      |      |      |      |    |    |     |     |     |     |                |               |               |               |               |               |          |                |               |               |              |               |               |          |            |            |                             |    S<C, D0, D1>|
///###### RCR
        // kernel 1: 256->32x128x128 
        // <      Row,      Col, DsLayout, ELayout, A0DataType, B0DataType, DsDataType, EDataType, AccDataType, CShuffleDataType,  AElementOp,  BElementOp, CDEElementOp,       GemmSpec,   256,   32,   128,    128,  16,  16,  32,   32,    1,    1,     S<8, 32, 1>,     S<1, 0, 2>,    S<1, 0, 2>,               2,             16,             16,          0,     S<8, 32, 1>,    S<1, 0, 2>,     S<1, 0, 2>,             2,              16,             16,          0,          1,           1,               S<1, 32, 1, 8>,      S<8, 8, 1>,  ck::BlockGemmPipelineScheduler::Interwave, ck::BlockGemmPipelineVersion::v1, F16>;
        // <      Row,      Col, DsLayout, ELayout, A0DataType, B0DataType, DsDataType, EDataType, AccDataType, CShuffleDataType,  AElementOp,  BElementOp, CDEElementOp,       GemmSpec,   256,   32,   128,    256,  16,  16,  32,   32,    1,    1,     S<16, 16, 1>,     S<1, 0, 2>,    S<1, 0, 2>,               2,             16,             16,          0,     S<16, 16, 1>,    S<1, 0, 2>,     S<1, 0, 2>,             2,              16,             16,          0,          1,           1,               S<1, 32, 1, 8>,      S<8, 8, 1>,  ck::BlockGemmPipelineScheduler::Intrawave, ck::BlockGemmPipelineVersion::v3, F16>;
        <      Row,      Col, DsLayout, ELayout, A0DataType, B0DataType, DsDataType, EDataType, AccDataType, CShuffleDataType,
               AElementOp,  BElementOp, CDEElementOp,       GemmSpec,   256,
               32,   128,    128,
               8,   8,
               32,   32,
               1,    1,
               S<8, 32, 1>, S<1, 0, 2>, S<1, 0, 2>, 2, 8, 8, 0,
               S<8, 32, 1>, S<1, 0, 2>, S<1, 0, 2>, 2, 8, 8, 0,
               //    CShuffle|    CShuffle| CBlockTransferClusterLengths|  CBlockTransfer|
               //    MXdlPerWave| NXdlPerWave|         _MBlock_MWaveMPerXdl| ScalarPerVector|
                //  PerShuffle|  PerShuffle|         _NBlock_NWaveNPerXdl|   _NWaveNPerXdl|
               1,    1,   S<1, 32, 1, 8>, S<8, 8, 1>,
               ck::BlockGemmPipelineScheduler::Intrawave, ck::BlockGemmPipelineVersion::v1, F16>;
        // kernel 2: 128->32x128x128
        //  <      Row,      Col, DsLayout, ELayout, A0DataType, B0DataType, DsDataType, EDataType, AccDataType, CShuffleDataType,  AElementOp,  BElementOp, CDEElementOp,       GemmSpec,   128,   32,   128,    128,  16,  16,  32,   32,    1,    2,     S<8, 16, 1>,     S<1, 0, 2>,    S<1, 0, 2>,               2,             16,             16,          0,     S<8, 16, 1>,    S<1, 0, 2>,     S<1, 0, 2>,             2,              16,             16,          0,          1,           1,               S<1, 16, 1, 8>,      S<8, 8, 1>,  ck::BlockGemmPipelineScheduler::Interwave, ck::BlockGemmPipelineVersion::v1, F16>;

    // clang-format on

    auto a_element_op = AElementOp{};
    auto b_element_op = BElementOp{};
    auto cde_element_op = CDEElementOp{};

    constexpr ck::index_t NumDTensor = DsDataType::Size();

    constexpr auto I0 = ck::Number<0>{};

    // do GEMM
    auto device_op = DeviceOpInstance{};

    auto invoker = device_op.MakeInvoker();
    auto argument =
        device_op.MakeArgument(sorted_token_ids.data_ptr(),
                               sorted_expert_ids.data_ptr(),
                               hidden_states.data_ptr(),
                               w1.data_ptr(),
                               std::array<const void *, NumDTensor>{a1_scale.has_value() ? a1_scale.value().data_ptr() : nullptr,
                                                                    w1_scale.has_value() ? w1_scale.value().data_ptr() : nullptr},
                               out.data_ptr(),
                               tokens,
                               SORTED_SIZE,
                               N,
                               K,
                               StrideA,
                               StrideB,
                               std::array<ck::index_t, NumDTensor>{I0, I0},
                               StrideE,
                               KBatch,
                               a_element_op,
                               b_element_op,
                               cde_element_op);

    if (!device_op.IsSupportedArgument(argument))
    {
        throw std::runtime_error(
            "wrong! device_gemm with the specified compilation parameters does "
            "not support this GEMM problem");
    }
    invoker.Run(argument, StreamConfig{at::cuda::getCurrentCUDAStream().stream()});
}

void ck_moe_stage2(torch::Tensor &hidden_states,          // [m, k], input token
    torch::Tensor &w1,                     // [e, n, k]/[e, 2*n, k], pre-shuffle([e, nr, kr, w])
    torch::Tensor &w2,                     // [expert, dim, inter_dim], pre-shuffle([e, nr, kr, w])
    torch::Tensor &sorted_token_ids,       // [max_num_tokens_padded]
    torch::Tensor &sorted_expert_ids,      // [max_num_m_blocks]
    torch::Tensor &out,                    // [max_num_tokens_padded, inter_dim]
    std::optional<torch::Tensor> w2_scale, // [e, 1, n], gate(up) scale
    std::optional<torch::Tensor> a1_scale  // [m, 1], token scale
)
{
int tokens = hidden_states.size(0);
int SORTED_SIZE = out.size(0);
int N = w2.size(2);
int K = w1.size(2);

// ~~~~~~~~~~~~~~~~~~~~~~~~following start with ck things
ck::index_t StrideA = K;
ck::index_t StrideB = K;
ck::index_t StrideD = 0;
ck::index_t StrideE = N;
ck::index_t KBatch = 1;
using PassThrough = ck::tensor_operation::element_wise::PassThrough;

using AElementOp = PassThrough;
using BElementOp = PassThrough;
using CDEElementOp = MultiplyMultiply;

static constexpr auto GemmSpec = ck::tensor_operation::device::GemmSpecialization::Default;

// using DeviceOpInstance = ck::tensor_operation::device::DeviceGemmMultiD_Xdl_CShuffle_V3
using DeviceOpInstance = ck::tensor_operation::device::DeviceGemmMultiD_Xdl_CShuffle_V3_BPreshuffle
// clang-format off
///######|  ALayout|  BLayout| DsLayout| ELayout|      AData|      BData|     DsData|     EData|     AccData|         CShuffle|           A|           B|          CDE|           GEMM| Block|  MPer|  NPer|  KPer| AK1| BK1| MPer| NPer| MXdl| NXdl|  ABlockTransfer| ABlockTransfer| ABlockTransfer| ABlockTransfer| ABlockTransfer| ABlockTransfer| ABlockLds|  BBlockTransfer| BBlockTransfer| BBlockTransfer| BlockTransfer| BBlockTransfer| BBlockTransfer| BBlockLds|    CShuffle|    CShuffle| CBlockTransferClusterLengths|  CBlockTransfer|
///######|         |         |         |        |       Type|       Type|       Type|      Type|        Type|         DataType| Elementwise| Elementwise|  Elementwise| Spacialization|  Size| Block| Block| Block|    |    |  XDL|  XDL|  Per|  Per|   ThreadCluster|  ThreadCluster| SrcAccessOrder|   SrcVectorDim|      SrcScalar|      DstScalar| AddExtraM|   ThreadCluster|  ThreadCluster| SrcAccessOrder|  SrcVectorDim|      SrcScalar|      DstScalar| AddExtraN| MXdlPerWave| NXdlPerWave|         _MBlock_MWaveMPerXdl| ScalarPerVector|
///######|         |         |         |        |           |           |           |          |            |                 |   Operation|   Operation|    Operation|               |      |      |      |      |    |    |     |     | Wave| Wave| Lengths_K0_M_K1|   ArrangeOrder|               |               |      PerVector|   PerVector_K1|          | Lengths_K0_N_K1|   ArrangeOrder|               |              |      PerVector|   PerVector_K1|          |  PerShuffle|  PerShuffle|         _NBlock_NWaveNPerXdl|   _NWaveNPerXdl|
///######|         |         |         |        |           |           |           |          |            |                 |            |            |             |               |      |      |      |      |    |    |     |     |     |     |                |               |               |               |               |               |          |                |               |               |              |               |               |          |            |            |                             |    S<C, D0, D1>|
///###### RCR
// kernel 1: 256->32x128x128 
// <      Row,      Col, DsLayout, ELayout, A0DataType, B0DataType, DsDataType, EDataType, AccDataType, CShuffleDataType,  AElementOp,  BElementOp, CDEElementOp,       GemmSpec,   256,   32,   128,    128,  16,  16,  32,   32,    1,    1,     S<8, 32, 1>,     S<1, 0, 2>,    S<1, 0, 2>,               2,             16,             16,          0,     S<8, 32, 1>,    S<1, 0, 2>,     S<1, 0, 2>,             2,              16,             16,          0,          1,           1,               S<1, 32, 1, 8>,      S<8, 8, 1>,  ck::BlockGemmPipelineScheduler::Interwave, ck::BlockGemmPipelineVersion::v1, F16>;
// <      Row,      Col, DsLayout, ELayout, A0DataType, B0DataType, DsDataType, EDataType, AccDataType, CShuffleDataType,  AElementOp,  BElementOp, CDEElementOp,       GemmSpec,   256,   32,   128,    256,  16,  16,  32,   32,    1,    1,     S<16, 16, 1>,     S<1, 0, 2>,    S<1, 0, 2>,               2,             16,             16,          0,     S<16, 16, 1>,    S<1, 0, 2>,     S<1, 0, 2>,             2,              16,             16,          0,          1,           1,               S<1, 32, 1, 8>,      S<8, 8, 1>,  ck::BlockGemmPipelineScheduler::Intrawave, ck::BlockGemmPipelineVersion::v3, F16>;
<      Row,      Col, DsLayout, ELayout, A0DataType, B0DataType, DsDataType, EDataType, AccDataType, CShuffleDataType,
AElementOp,  BElementOp, CDEElementOp,       GemmSpec,   256,
32,   128,    128,
8,   8,
32,   32,
1,    1,
S<8, 32, 1>, S<1, 0, 2>, S<1, 0, 2>, 2, 8, 8, 0,
S<8, 32, 1>, S<1, 0, 2>, S<1, 0, 2>, 2, 8, 8, 0,
//    CShuffle|    CShuffle| CBlockTransferClusterLengths|  CBlockTransfer|
//    MXdlPerWave| NXdlPerWave|         _MBlock_MWaveMPerXdl| ScalarPerVector|
 //  PerShuffle|  PerShuffle|         _NBlock_NWaveNPerXdl|   _NWaveNPerXdl|
1,    1,   S<1, 32, 1, 8>, S<8, 8, 1>,
ck::BlockGemmPipelineScheduler::Intrawave, ck::BlockGemmPipelineVersion::v1, F16>;
// kernel 2: 128->32x128x128
//  <      Row,      Col, DsLayout, ELayout, A0DataType, B0DataType, DsDataType, EDataType, AccDataType, CShuffleDataType,  AElementOp,  BElementOp, CDEElementOp,       GemmSpec,   128,   32,   128,    128,  16,  16,  32,   32,    1,    2,     S<8, 16, 1>,     S<1, 0, 2>,    S<1, 0, 2>,               2,             16,             16,          0,     S<8, 16, 1>,    S<1, 0, 2>,     S<1, 0, 2>,             2,              16,             16,          0,          1,           1,               S<1, 16, 1, 8>,      S<8, 8, 1>,  ck::BlockGemmPipelineScheduler::Interwave, ck::BlockGemmPipelineVersion::v1, F16>;

// clang-format on

auto a_element_op = AElementOp{};
auto b_element_op = BElementOp{};
auto cde_element_op = CDEElementOp{};

constexpr ck::index_t NumDTensor = DsDataType::Size();

constexpr auto I0 = ck::Number<0>{};

// do GEMM
auto device_op = DeviceOpInstance{};

auto invoker = device_op.MakeInvoker();
auto argument =
device_op.MakeArgument(sorted_token_ids.data_ptr(),
                sorted_expert_ids.data_ptr(),
                hidden_states.data_ptr(),
                w2.data_ptr(),
                std::array<const void *, NumDTensor>{a1_scale.has_value() ? a1_scale.value().data_ptr() : nullptr,
                                                     w2_scale.has_value() ? w2_scale.value().data_ptr() : nullptr},
                out.data_ptr(),
                tokens,
                SORTED_SIZE,
                N,
                K,
                StrideA,
                StrideB,
                std::array<ck::index_t, NumDTensor>{I0, I0},
                StrideE,
                KBatch,
                a_element_op,
                b_element_op,
                cde_element_op);

if (!device_op.IsSupportedArgument(argument))
{
throw std::runtime_error(
"wrong! device_gemm with the specified compilation parameters does "
"not support this GEMM problem");
}
invoker.Run(argument, StreamConfig{at::cuda::getCurrentCUDAStream().stream()});
}